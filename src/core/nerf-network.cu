#include "hip/hip_runtime.h"
// This code was adapted from nerfstudio (Copyright 2022 The Nerfstudio Team. All rights reserved.)
// https://github.com/nerfstudio-project/nerfstudio/blob/main/nerfstudio/fields/instant_ngp_field.py
// Please see LICENSES/nerfstudio-project_nerfstudio.md for license details.

#include <json/json.hpp>
#include <math.h>
#include <tiny-cuda-nn/common.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

#include "../utils/gpu-image.cuh"
#include "../utils/parallel-utils.cuh"
#include "../utils/training-network-kernels.cuh"
#include "nerf-network.cuh"

using namespace tcnn;
using namespace nrc;
using json = nlohmann::json;


#if TCNN_HALF_PRECISION
    constexpr float LOSS_SCALE = 128.0f;
#else
    constexpr float LOSS_SCALE = 1.0f;
#endif


// Constructor

NerfNetwork::NerfNetwork(
	const int& device_id,
	const float& aabb_size
)
	: network_ws(device_id)
	, params_ws(device_id)
{
	this->aabb_size = aabb_size;

	// These values are from the Instant-NGP paper, page 4. "Multiresolution Hash Encoding"
	double n_levels = 16.0;
	double N_min = 16.0;
	double N_max = 524288.0;
	double b = exp((log(N_max) - log(N_min)) / (n_levels - 1.0));

	// These network configurations were adapted from nerfstudio
	
	// Create the Direction Encoding
	json direction_encoding_config = {
		{"otype", "SphericalHarmonics"},
		{"degree", 4},
	};

	direction_encoding.reset(
		create_encoding<network_precision_t>(3, direction_encoding_config)
	);

	// Create the Density MLP

	json density_encoding_config = {
		{"otype", "HashGrid"},
		{"n_levels", 16},
		{"n_features_per_level", 2},
		{"log2_hashmap_size", 19},
		{"base_resolution", 16},
		{"per_level_scale", b},
		// used by recommendation of Müller et al (instant-NGP paper, page 13 "Smooth Interpolation")
		{"interpolation", "Smoothstep"},
	};

	json density_network_config = {
		{"otype", "FullyFusedMLP"},
		{"activation", "ReLU"},
		{"output_activation", "None"},
		{"n_neurons", 64},
		{"n_hidden_layers", 1},
	};

	density_network.reset(
		new NetworkWithInputEncoding<network_precision_t>(
			3,	// input dims
			16, // output dims
			density_encoding_config,
			density_network_config
		)
	);

	// Create the Color MLP

	uint32_t color_network_in_dim = direction_encoding->padded_output_width() + density_network->padded_output_width();

	const json color_network_config = {
		{"otype", "FullyFusedMLP"},
		{"activation", "ReLU"},
		{"output_activation", "Sigmoid"},
		{"n_neurons", 64},
		{"n_hidden_layers", 2},
		{"n_input_dims", color_network_in_dim},
		{"n_output_dims", 3},
	};

	color_network.reset(
		create_network<network_precision_t>(color_network_config)
	);
}

// initialize params and gradients for the networks (I have no idea if this is correct)
void NerfNetwork::prepare_for_training(const hipStream_t& stream) {

	size_t rng_seed = 72791;
	pcg32 rng(rng_seed);

	// initialize params
	params_ws.enlarge(
		stream,
		density_network->n_params(),
		color_network->n_params()
	);
	
	density_network->initialize_params(rng, params_ws.density_network_params_fp);
	color_network->initialize_params(rng, params_ws.color_network_params_fp);

	// initialize_params only initializes full precision params, need to copy to half precision

	copy_and_cast<network_precision_t, float>(
		stream,
		density_network->n_params(),
		params_ws.density_network_params_hp,
		params_ws.density_network_params_fp
	);

	copy_and_cast<network_precision_t, float>(
		stream,
		color_network->n_params(),
		params_ws.color_network_params_hp,
		params_ws.color_network_params_fp
	);

	// assign params pointers

	density_network->set_params(
		params_ws.density_network_params_hp,
		params_ws.density_network_params_hp,
		params_ws.density_network_gradients_hp
	);

	color_network->set_params(
		params_ws.color_network_params_hp,
		params_ws.color_network_params_hp,
		params_ws.color_network_gradients_hp
	);

	// initialize optimizers
	
	json optimizer_config = {
		{"otype", "Adam"},
		{"learning_rate", 1e-2},
		{"epsilon", 1e-15},
		{"l2_reg", 1e-6}
	};

	optimizer.reset(
		new NGPAdamOptimizer<network_precision_t>(optimizer_config)
	);

	size_t n_params = density_network->n_params() + color_network->n_params();
	uint32_t n_grid_params = density_network->encoding()->n_params();
	optimizer->allocate(n_params, {{n_grid_params, 1}});

	// flag for training enabled
	can_train = true;
}

void NerfNetwork::train(
	const hipStream_t& stream,
	const uint32_t& batch_size,
	const uint32_t& n_rays,
	const uint32_t& n_samples,
	uint32_t* ray_steps,
	uint32_t* ray_offset,
	float* pos_batch,
	float* dir_batch,
	float* dt_batch,
	float* target_rgba,
	network_precision_t* concat_buffer,
	network_precision_t* output_buffer
) {
	
	enlarge_workspace_if_needed(stream, batch_size);

	// Forward
	auto fwd_ctx = forward(
		stream,
		batch_size,
		n_rays,
		n_samples,
		ray_steps,
		ray_offset,
		target_rgba,
		pos_batch,
		dir_batch,
		dt_batch,
		concat_buffer,
		output_buffer
	);

	// Loss
	const float loss = calculate_loss(
		stream,
		batch_size,
		n_rays
	);

	printf(
		"Loss: %f / # Rays: %u\n",
		loss,
		n_rays
	);

	// Backward
	backward(
		stream,
		fwd_ctx,
		n_rays,
		n_samples,
		batch_size,
		ray_steps,
		ray_offset,
		concat_buffer,
		output_buffer,
		pos_batch,
		dir_batch,
		dt_batch,
		target_rgba
	);

	// Optimizer
	optimizer_step(stream);
}

void NerfNetwork::inference(
	const hipStream_t& stream,
	const uint32_t& batch_size,
	float* pos_batch,
	float* dir_batch,
	// density network output must have space available for (color_network->input_width() * batch_size) elements of type network_precision_t
	network_precision_t* concat_buffer,
	// color network output must have space available for (color_network->padded_output_width() * batch_size) elements of type network_precision_t
	network_precision_t* output_buffer,
	// if this flag is false, we only run inference on the density network
	const bool& use_color_network
) {
	// Inference (density network)
	GPUMatrixDynamic density_network_input_matrix(
		pos_batch,
		density_network->input_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	GPUMatrixDynamic density_network_output_matrix(
		concat_buffer,
		density_network->padded_output_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	density_network->inference_mixed_precision(
		stream,
		density_network_input_matrix,
		density_network_output_matrix
	);

	if (use_color_network) {
		// Inference (direction encoding)
		network_precision_t* direction_encoding_output = concat_buffer + density_network->padded_output_width() * batch_size;

		GPUMatrixDynamic direction_encoding_input_matrix(
			dir_batch,
			direction_encoding->input_width(),
			batch_size,
			MatrixLayout::RowMajor
		);

		GPUMatrixDynamic direction_encoding_output_matrix(
			direction_encoding_output,
			direction_encoding->padded_output_width(),
			batch_size,
			MatrixLayout::RowMajor
		);

		direction_encoding->inference_mixed_precision(
			stream,
			direction_encoding_input_matrix,
			direction_encoding_output_matrix
		);

		// Inference (color network)
		GPUMatrixDynamic color_network_input_matrix(
			density_network_output_matrix.data(),
			color_network->input_width(),
			batch_size,
			MatrixLayout::RowMajor
		);

		GPUMatrixDynamic color_network_output_matrix(
			output_buffer,
			color_network->padded_output_width(),
			batch_size,
			MatrixLayout::RowMajor
		);

		color_network->inference_mixed_precision(
			stream,
			color_network_input_matrix,
			color_network_output_matrix
		);
	}

	// for inference we just overwrite the color network's alpha channel with activated sigma data
	density_to_sigma_forward_kernel<<<n_blocks_linear(batch_size), tcnn::n_threads_linear, 0, stream>>>(
		batch_size,
		concat_buffer,
		output_buffer + 3 * batch_size
	);
}

std::unique_ptr<NerfNetwork::ForwardContext> NerfNetwork::forward(
	const hipStream_t& stream,
	const uint32_t& batch_size,
	const uint32_t& n_rays,
	const uint32_t& n_samples,
	const uint32_t* ray_steps,
	const uint32_t* ray_offset,
	const float* target_rgba,
	float* pos_batch,
	float* dir_batch,
	float* dt_batch,
	network_precision_t* concat_buffer,
	network_precision_t* output_buffer
) {
	auto fwd_ctx = std::make_unique<ForwardContext>();

	// Forward pass on density network (with multiresolution hash encoding built in!)

	fwd_ctx->density_network_input_matrix = GPUMatrixDynamic(
		pos_batch,								// density network takes the sample positions as input
		density_network->input_width(),			// rows
		batch_size,								// cols
		MatrixLayout::RowMajor
	);

	// Here we make the output of the density network a pointer to the first half of the color network's input buffer.
	fwd_ctx->density_network_output_matrix = GPUMatrixDynamic(
		concat_buffer,				 			// density network output = color network input
		density_network->output_width(), 		// rows
		batch_size,								// cols
		MatrixLayout::RowMajor
	);

	fwd_ctx->density_ctx = density_network->forward(
		stream,
		fwd_ctx->density_network_input_matrix,
		&fwd_ctx->density_network_output_matrix,
		false,
		true // prepare_input_gradients must be `true` otherwise backward() fails (forward->dy_dx is not defined)
	);

	// Encode directions (dir_batch)
	// Direction encoding gets concatenated with density_network_output (which will just be the second half of concat_buffer)

	network_precision_t* direction_encoding_output = concat_buffer + density_network->padded_output_width() * batch_size;

	fwd_ctx->direction_encoding_input_matrix = GPUMatrixDynamic(
		dir_batch,									// pointer to source data
		direction_encoding->input_width(),			// rows
		batch_size,									// cols
		MatrixLayout::RowMajor
	);

	fwd_ctx->direction_encoding_output_matrix = GPUMatrixDynamic(
		direction_encoding_output,					// pointer to destination data
		direction_encoding->padded_output_width(),	// rows
		batch_size,									// cols
		MatrixLayout::RowMajor
	);

	direction_encoding->forward(
		stream,
		fwd_ctx->direction_encoding_input_matrix,
		&fwd_ctx->direction_encoding_output_matrix
	);

	// Perform the forward pass on the color network

	fwd_ctx->color_network_input_matrix = GPUMatrixDynamic(
		concat_buffer,							// pointer to source data
		color_network->input_width(),			// matrix rows
		batch_size,								// matrix columns
		MatrixLayout::RowMajor
	);

	fwd_ctx->color_network_output_matrix = GPUMatrixDynamic(
		output_buffer,							// pointer to destination data
		color_network->padded_output_width(),	// matrix rows
		batch_size,								// matrix columns
		MatrixLayout::RowMajor
	);

	fwd_ctx->color_ctx = color_network->forward(
		stream,
		fwd_ctx->color_network_input_matrix,
		&fwd_ctx->color_network_output_matrix,
		false,
		true // prepare_input_gradients
	);
	
	// Zero out transmittance
	hipMemsetAsync(network_ws.trans_buf, 0, batch_size * sizeof(float), stream);

	// Continue forward with custom operators
	density_to_sigma_forward_kernel_fused<<<n_blocks_linear(n_samples), n_threads_linear, 0, stream>>>(
		n_samples,
		concat_buffer,
		network_ws.sigma_buf,
        dt_batch,
        network_ws.alpha_buf
	);

	sigma_to_ray_rgba_forward_kernel<<<n_blocks_linear(n_rays), n_threads_linear, 0, stream>>>(
		n_rays,
		batch_size,
		ray_steps,
		ray_offset,
		output_buffer,
		network_ws.alpha_buf,
		network_ws.ray_rgba
	);

	ray_rgba_to_loss_forward_kernel<<<n_blocks_linear(batch_size), n_threads_linear, 0, stream>>>(
		n_rays,
		batch_size,
		network_ws.ray_rgba,
		target_rgba,
		network_ws.loss_buf
	);

	return fwd_ctx;
}

float NerfNetwork::calculate_loss(
	const hipStream_t& stream,
	const uint32_t& batch_size,
	const uint32_t& n_rays
) {
	// Add all loss values together
	thrust::device_ptr<float> loss_buffer_ptr(network_ws.loss_buf);

	return (1.0f / (float)n_rays) * thrust::reduce(
		thrust::cuda::par_nosync.on(stream),
		loss_buffer_ptr,
		loss_buffer_ptr + 4 * batch_size,
		0.0f,
		thrust::plus<float>()
	);
}

void NerfNetwork::backward(
	const hipStream_t& stream,
	const std::unique_ptr<NerfNetwork::ForwardContext>& fwd_ctx,
	const uint32_t& n_rays,
	const uint32_t& n_samples,
	const uint32_t& batch_size,
	const uint32_t* ray_steps,
	const uint32_t* ray_offset,
	const tcnn::network_precision_t* network_density,
	const tcnn::network_precision_t* network_color,
	float* pos_batch,
	float* dir_batch,
	float* dt_batch,
	float* target_rgba
) {
	// zero out previous gradients
	hipMemsetAsync(network_ws.grad_dL_dR, 0, 4 * batch_size * sizeof(float), stream);
	hipMemsetAsync(network_ws.grad_dL_dcolor, 0, 3 * batch_size * sizeof(float), stream);
	hipMemsetAsync(network_ws.grad_dL_dsigma, 0, batch_size * sizeof(float), stream);

	// Backpropagate loss
	ray_rgba_to_loss_backward_kernel<<<n_blocks_linear(n_rays), n_threads_linear, 0, stream>>>(
		n_rays,
		batch_size,
		1.0f / (2.0f * (float)n_rays),
		network_ws.ray_rgba,
		target_rgba,
		network_ws.grad_dL_dR
	);

	sigma_to_ray_rgba_backward_kernel<<<n_blocks_linear(n_rays), n_threads_linear, 0, stream>>>(
		n_rays,
		batch_size,
		ray_steps,
		ray_offset,
		dt_batch,
		network_ws.alpha_buf,
		network_color,
		network_ws.ray_rgba,
		network_ws.grad_dL_dR,
		network_ws.grad_dL_dsigma,
		network_ws.grad_dL_dcolor
	);

	density_to_sigma_backward_kernel<<<n_blocks_linear(n_samples), n_threads_linear, 0, stream>>>(
		n_samples,
		network_ws.sigma_buf,
		network_ws.grad_dL_dsigma,
		network_ws.grad_dL_ddensity
	);

	// Backpropagate through the color network
	GPUMatrixDynamic color_network_dL_doutput_matrix(
		network_ws.color_network_dL_doutput,
		color_network->padded_output_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	// need to clear & populate color network matrix
	color_network_dL_doutput_matrix.memset_async(stream, 0);

	copy_gradients_kernel<3, false><<<n_blocks_linear(n_samples), n_threads_linear, 0, stream>>>(
		n_samples,
		batch_size,
		LOSS_SCALE,
		network_ws.grad_dL_dcolor,
		color_network_dL_doutput_matrix.data()
	);

	GPUMatrixDynamic color_network_dL_dinput_matrix(
		network_ws.color_network_dL_dinput,
		color_network->input_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	color_network->backward(
		stream,
		*fwd_ctx->color_ctx,
		fwd_ctx->color_network_input_matrix,
		fwd_ctx->color_network_output_matrix,
		color_network_dL_doutput_matrix,
		&color_network_dL_dinput_matrix
	);

	// Backpropagate through the density network
	GPUMatrixDynamic density_network_dL_dinput_matrix(
		network_ws.density_network_dL_dinput,
		density_network->input_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	// Construct a dL_dinput matrix of the correct size
	// color_network_dL_dinput_matrix is too large since it is the concatenation of density's outputs and encoded directions

	GPUMatrixDynamic density_network_dL_doutput_matrix(
		color_network_dL_dinput_matrix.data(),
		density_network->padded_output_width(),
		batch_size,
		MatrixLayout::RowMajor
	);

	// We need to add dL/ddensity to dL/doutput before backpropagating
	copy_gradients_kernel<1, true><<<n_blocks_linear(n_samples), n_threads_linear, 0, stream>>>(
		n_samples,
		batch_size,
		LOSS_SCALE,
		network_ws.grad_dL_ddensity,
		density_network_dL_doutput_matrix.data()
	);

	density_network->backward(
		stream,
		*fwd_ctx->density_ctx,
		fwd_ctx->density_network_input_matrix,
		fwd_ctx->density_network_output_matrix,
		density_network_dL_doutput_matrix,
		&density_network_dL_dinput_matrix
	);

}

void NerfNetwork::optimizer_step(const hipStream_t& stream) {

	optimizer->step(
		stream,
		LOSS_SCALE,
		params_ws.params_fp,
		params_ws.params_hp,
		params_ws.gradients_hp
	);

}

// Only enlarge buffers needed for inference
void NerfNetwork::enlarge_workspace_if_needed(const hipStream_t& stream, const uint32_t& batch_size) {
	if (batch_size <= this->batch_size) {
		return;
	}

	network_ws.enlarge(
		stream,
		batch_size,
		density_network->input_width(),
		density_network->padded_output_width(),
		direction_encoding->input_width(),
		direction_encoding->padded_output_width(),
		color_network->input_width(),
		color_network->padded_output_width()
	);

	this->batch_size = batch_size;
}
